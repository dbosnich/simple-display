#include "hip/hip_runtime.h"
//--------------------------------------------------------------
// Copyright (c) David Bosnich <david.bosnich.public@gmail.com>
//
// This code is licensed under the MIT License, a copy of which
// can be found in the license.txt file included at the root of
// this distribution, or at https://opensource.org/licenses/MIT
//--------------------------------------------------------------

#include <simple/display/buffer.h>

#include <hip/hip_runtime.h>

using namespace Simple::Display;
using namespace std;

__constant__ float COLORS_FLOAT[4][4];
__constant__ uint8_t COLORS_UINT8[4][4];
__constant__ uint16_t COLORS_UINT16[4][4];

//--------------------------------------------------------------
template <typename BufferType>
__global__ void CycleColorsKernel(BufferType* a_pixelBuffer,
                                  uint32_t a_pixelWidth,
                                  uint32_t a_pixelHeight,
                                  uint32_t a_numChannels,
                                  uint32_t a_secondsElapsed)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= a_pixelWidth || y >= a_pixelHeight)
    {
        return;
    }

    const uint32_t topLeftIndex = a_secondsElapsed % 4;
    const int topRightIndex = (topLeftIndex == 3) ? 0 : topLeftIndex + 1;
    const int bottomLeftIndex = (topRightIndex == 3) ? 0 : topRightIndex + 1;
    const int bottomRightIndex = (bottomLeftIndex == 3) ? 0 : bottomLeftIndex + 1;

    const BufferType* colorTopLeft = nullptr;
    const BufferType* colorTopRight = nullptr;
    const BufferType* colorBottomLeft = nullptr;
    const BufferType* colorBottomRight = nullptr;
    if constexpr (std::is_same<BufferType, float>::value)
    {
        colorTopLeft = COLORS_FLOAT[topLeftIndex];
        colorTopRight = COLORS_FLOAT[topRightIndex];
        colorBottomLeft = COLORS_FLOAT[bottomLeftIndex];
        colorBottomRight = COLORS_FLOAT[bottomRightIndex];
    }
    else if constexpr (std::is_same<BufferType, uint8_t>::value)
    {
        colorTopLeft = COLORS_UINT8[topLeftIndex];
        colorTopRight = COLORS_UINT8[topRightIndex];
        colorBottomLeft = COLORS_UINT8[bottomLeftIndex];
        colorBottomRight = COLORS_UINT8[bottomRightIndex];
    }
    else if constexpr (std::is_same<BufferType, uint16_t>::value)
    {
        colorTopLeft = COLORS_UINT16[topLeftIndex];
        colorTopRight = COLORS_UINT16[topRightIndex];
        colorBottomLeft = COLORS_UINT16[bottomLeftIndex];
        colorBottomRight = COLORS_UINT16[bottomRightIndex];
    }
    else
    {
        static_assert(!std::is_same<BufferType, BufferType>::value, "Unsupported type");
    }

    const uint32_t quadrant = (x > (a_pixelWidth / 2)) +
                              (2 * (y > (a_pixelHeight / 2)));
    const BufferType* color = colorTopLeft;
    switch (quadrant)
    {
        case 0: color = colorBottomLeft; break;
        case 1: color = colorBottomRight; break;
        case 2: color = colorTopLeft; break;
        case 3: color = colorTopRight; break;
    }

    const uint32_t i = (x * a_numChannels) +
                       (y * a_pixelWidth * a_numChannels);
    for (uint32_t z = 0; z < a_numChannels; ++z)
    {
        a_pixelBuffer[i + z] = color[z];
    }
}

//--------------------------------------------------------------
template <typename BufferType>
extern void CycleColorsCuda(const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    BufferType* pixelBuffer = a_buffer.GetData<BufferType, Buffer::Interop::CUDA>();
    if (!pixelBuffer)
    {
        return;
    }

    const uint32_t pixelWidth = a_buffer.GetWidth();
    const uint32_t pixelHeight = a_buffer.GetHeight();
    const uint32_t numChannels = Buffer::ChannelsPerPixel(a_buffer.GetFormat());

    dim3 blockDim(16, 16);
    dim3 gridDim((pixelWidth + blockDim.x - 1) / blockDim.x,
                 (pixelHeight + blockDim.y - 1) / blockDim.y);
    CycleColorsKernel<<<gridDim, blockDim>>>(pixelBuffer,
                                             pixelWidth,
                                             pixelHeight,
                                             numChannels,
                                             (int)a_secondsElapsed);
}

//--------------------------------------------------------------
extern void CycleColorsCuda(const float a_colors[4][4],
                            const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    hipMemcpyToSymbol(HIP_SYMBOL(COLORS_FLOAT), a_colors, 16 * sizeof(float));
    CycleColorsCuda<float>(a_buffer, a_secondsElapsed);
}

//--------------------------------------------------------------
extern void CycleColorsCuda(const uint8_t a_colors[4][4],
                            const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    hipMemcpyToSymbol(HIP_SYMBOL(COLORS_UINT8), a_colors, 16 * sizeof(uint8_t));
    CycleColorsCuda<uint8_t>(a_buffer, a_secondsElapsed);
}

//--------------------------------------------------------------
extern void CycleColorsCuda(const uint16_t a_colors[4][4],
                            const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    hipMemcpyToSymbol(HIP_SYMBOL(COLORS_UINT16), a_colors, 16 * sizeof(uint16_t));
    CycleColorsCuda<uint16_t>(a_buffer, a_secondsElapsed);
}
