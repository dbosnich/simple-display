#include "hip/hip_runtime.h"
//--------------------------------------------------------------
// Copyright (c) David Bosnich <david.bosnich.public@gmail.com>
//
// This code is licensed under the MIT License, a copy of which
// can be found in the license.txt file included at the root of
// this distribution, or at https://opensource.org/licenses/MIT
//--------------------------------------------------------------

#include <simple/display/buffer.h>

#include <hip/hip_runtime.h>

using namespace Simple::Display;
using namespace std;

template <typename DataType, uint32_t ChannelsPerPixel, uint32_t NumColors>
__constant__ DataType COLORS[NumColors][ChannelsPerPixel];

//--------------------------------------------------------------
template<typename DataType, uint32_t ChannelsPerPixel, uint32_t NumColors>
__global__ void CycleColorsKernel(DataType* a_bufferData,
                                  uint32_t a_bufferWidth,
                                  uint32_t a_bufferHeight,
                                  uint32_t a_secondsElapsed)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= a_bufferWidth || y >= a_bufferHeight)
    {
        return;
    }

    const uint32_t topLeftIndex = a_secondsElapsed % NumColors;
    const uint32_t topRightIndex = (topLeftIndex == NumColors - 1) ? 0 : min(topLeftIndex + 1, NumColors);
    const uint32_t bottomLeftIndex = (topRightIndex == NumColors - 1) ? 0 : min(topRightIndex + 1, NumColors);
    const uint32_t bottomRightIndex = (bottomLeftIndex == NumColors - 1) ? 0 : min(bottomLeftIndex + 1, NumColors);

    const DataType* colorTopLeft = COLORS<DataType, ChannelsPerPixel, NumColors>[topLeftIndex];
    const DataType* colorTopRight = COLORS<DataType, ChannelsPerPixel, NumColors>[topRightIndex];
    const DataType* colorBottomLeft = COLORS<DataType, ChannelsPerPixel, NumColors>[bottomLeftIndex];
    const DataType* colorBottomRight = COLORS<DataType, ChannelsPerPixel, NumColors>[bottomRightIndex];

    const uint32_t quadrant = (x > (a_bufferWidth / 2)) +
                              (2 * (y > (a_bufferHeight / 2)));
    const DataType* color = colorTopLeft;
    switch (quadrant)
    {
        case 0: color = colorBottomLeft; break;
        case 1: color = colorBottomRight; break;
        case 2: color = colorTopLeft; break;
        case 3: color = colorTopRight; break;
    }

    const uint32_t i = (x * ChannelsPerPixel) +
                       (y * a_bufferWidth * ChannelsPerPixel);
    for (uint32_t z = 0; z < ChannelsPerPixel; ++z)
    {
        a_bufferData[i + z] = color[z];
    }
}

//--------------------------------------------------------------
template<typename DataType, uint32_t ChannelsPerPixel, uint32_t NumColors>
extern void CycleColorsCuda(const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    const uint32_t bufferWidth = a_buffer.GetWidth();
    const uint32_t bufferHeight = a_buffer.GetHeight();
    DataType* bufferData = a_buffer.GetData<DataType, Buffer::Interop::CUDA>();
    if (!bufferData || !bufferWidth || !bufferHeight || !bufferData)
    {
        return;
    }

    dim3 blockDim(16, 16);
    dim3 gridDim((bufferWidth + blockDim.x - 1) / blockDim.x,
                 (bufferHeight + blockDim.y - 1) / blockDim.y);
    CycleColorsKernel<DataType, ChannelsPerPixel, NumColors><<<gridDim, blockDim>>>(bufferData,
                                                                                    bufferWidth,
                                                                                    bufferHeight,
                                                                                    (uint32_t)a_secondsElapsed);
}

//--------------------------------------------------------------
template<typename DataType, uint32_t ChannelsPerPixel, uint32_t NumColors>
extern void CycleColorsCuda(const DataType a_colors[NumColors][ChannelsPerPixel],
                            const Buffer& a_buffer,
                            float a_secondsElapsed)
{
    // Note: This is not thread safe, but it avoids the calling
    // code needing CUDA specific code to pass a device pointer.
    hipMemcpyToSymbol(HIP_SYMBOL(COLORS<DataType), ChannelsPerPixel, NumColors>,
                       a_colors,
                       ChannelsPerPixel * NumColors * sizeof(DataType));
    CycleColorsCuda<DataType, ChannelsPerPixel, NumColors>(a_buffer, a_secondsElapsed);
}

//--------------------------------------------------------------
template void CycleColorsCuda<float, 4, 4>(const float a_colors[4][4],
                                           const Buffer& a_buffer,
                                           float a_secondsElapsed);

//--------------------------------------------------------------
template void CycleColorsCuda<uint8_t, 4, 4>(const uint8_t a_colors[4][4],
                                             const Buffer& a_buffer,
                                             float a_secondsElapsed);

//--------------------------------------------------------------
template void CycleColorsCuda<uint16_t, 4, 4>(const uint16_t a_colors[4][4],
                                              const Buffer& a_buffer,
                                              float a_secondsElapsed);
